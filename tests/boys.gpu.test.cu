#include "hip/hip_runtime.h"
// -*-c++-*-

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test.h"

#include "libintx/boys/gpu/chebyshev.h"
#include "libintx/gpu/api/api.h"

#include <hip/hip_cooperative_groups.h>


template<class Chebyshev>
__global__
void test(const Chebyshev chebyshev, double *ptr) {
  int bidx = blockIdx.x;
  int tidx = threadIdx.y + threadIdx.z*blockDim.y;
  ptr[bidx+tidx] = chebyshev.compute(bidx%128, tidx%8);
}

template<class Chebyshev>
__global__
void test_cooperative_groups(const Chebyshev chebyshev, double *ptr) {
  int bidx = blockIdx.x;
  int tidx = threadIdx.y + threadIdx.z*blockDim.y;
  auto p8 = cooperative_groups::tiled_partition<8>(cooperative_groups::this_thread_block());
  for (int i = 0; i < 8; ++i) {
    ptr[bidx+i] = chebyshev.compute(bidx%128, i, p8);
  }
}

template<int Order, int M, int Segments>
void test(int grid) {

  //dim3 block = { Order+1, 32/(Order+1), 4 };
  dim3 block = { 1, 8, 32 };

  typedef libintx::boys::gpu::Chebyshev<Order,M,117,Segments> Chebyshev;

  auto ptr = libintx::gpu::device::make_shared<double[]>(grid*block.y*block.z);

  auto chebyshev = Chebyshev();

  for (size_t i = 0; i < 5; ++i) {
    test<<<grid,block>>>(chebyshev, ptr.get());
    libintx::gpu::stream::synchronize();
    // test_cooperative_groups<<<grid,block>>>(chebyshev, ptr.get());
    // current_device.synchronize();
  }

  // cuda::launch(
  //   test<Chebyshev>,
  //   { grid, block },
  //   chebyshev,
  //   ptr.get()
  // );

  libintx::gpu::stream::synchronize();

}

TEST_CASE("chebyshev") { test<7,20,117*7>(100*128); }
TEST_CASE("chebyshev") { test<7,40,117*7>(100*128); }
TEST_CASE("chebyshev") { test<15,20,117*7>(100*128); }
TEST_CASE("chebyshev") { test<15,40,117*7>(100*128); }
